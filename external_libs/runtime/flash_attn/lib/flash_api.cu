#include "hip/hip_runtime.h"
#include "flash.h"
#include "flash_api.h"
#include "static_switch.h"
#include <iostream>
#include <algorithm>

extern "C" {
const char *VERSION = "2.5.3";
}

// for debug
void print_Qkv_params(Qkv_params &params) {
  std::cout << "q_batch_stride: " << params.q_batch_stride << std::endl;
  std::cout << "k_batch_stride: " << params.k_batch_stride << std::endl;
  std::cout << "v_batch_stride: " << params.v_batch_stride << std::endl;
  std::cout << "q_row_stride: " << params.q_row_stride << std::endl;
  std::cout << "k_row_stride: " << params.k_row_stride << std::endl;
  std::cout << "v_row_stride: " << params.v_row_stride << std::endl;
  std::cout << "q_head_stride: " << params.q_head_stride << std::endl;
  std::cout << "k_head_stride: " << params.k_head_stride << std::endl;
  std::cout << "v_head_stride: " << params.v_head_stride << std::endl;
  std::cout << "h: " << params.h << std::endl;
  std::cout << "h_k: " << params.h_k << std::endl;
  std::cout << "h_h_k_ratio: " << params.h_h_k_ratio << std::endl;
}

void print_Flash_fwd_params(Flash_fwd_params &params) {
  std::cout << "q_batch_stride: " << params.q_batch_stride << std::endl;
  std::cout << "k_batch_stride: " << params.k_batch_stride << std::endl;
  std::cout << "v_batch_stride: " << params.v_batch_stride << std::endl;
  std::cout << "q_row_stride: " << params.q_row_stride << std::endl;
  std::cout << "k_row_stride: " << params.k_row_stride << std::endl;
  std::cout << "v_row_stride: " << params.v_row_stride << std::endl;
  std::cout << "q_head_stride: " << params.q_head_stride << std::endl;
  std::cout << "k_head_stride: " << params.k_head_stride << std::endl;
  std::cout << "v_head_stride: " << params.v_head_stride << std::endl;
  std::cout << "h: " << params.h << std::endl;
  std::cout << "h_k: " << params.h_k << std::endl;
  std::cout << "h_h_k_ratio: " << params.h_h_k_ratio << std::endl;

  std::cout << "o_batch_stride: " << params.o_batch_stride << std::endl;
  std::cout << "o_row_stride: " << params.o_row_stride << std::endl;
  std::cout << "o_head_stride: " << params.o_head_stride << std::endl;
  std::cout << "b: " << params.b << std::endl;
  std::cout << "seqlen_q: " << params.seqlen_q << std::endl;
  std::cout << "seqlen_k: " << params.seqlen_k << std::endl;
  std::cout << "d: " << params.d << std::endl;
  std::cout << "seqlen_q_rounded: " << params.seqlen_q_rounded << std::endl;
  std::cout << "seqlen_k_rounded: " << params.seqlen_k_rounded << std::endl;
  std::cout << "d_rounded: " << params.d_rounded << std::endl;
  std::cout << "scale_softmax: " << params.scale_softmax << std::endl;
  std::cout << "scale_softmax_log2: " << params.scale_softmax_log2 << std::endl;
  std::cout << "p_dropout: " << params.p_dropout << std::endl;
  std::cout << "p_dropout_in_uint8_t: " << params.p_dropout_in_uint8_t
            << std::endl;
  std::cout << "rp_dropout: " << params.rp_dropout << std::endl;
  std::cout << "scale_softmax_rp_dropout: " << params.scale_softmax_rp_dropout
            << std::endl;
  std::cout << "is_bf16: " << params.is_bf16 << std::endl;
  std::cout << "is_causal: " << params.is_causal << std::endl;
}

void print_Flash_bwd_params(Flash_bwd_params &params) {
  std::cout << "q_batch_stride: " << params.q_batch_stride << std::endl;
  std::cout << "k_batch_stride: " << params.k_batch_stride << std::endl;
  std::cout << "v_batch_stride: " << params.v_batch_stride << std::endl;
  std::cout << "q_row_stride: " << params.q_row_stride << std::endl;
  std::cout << "k_row_stride: " << params.k_row_stride << std::endl;
  std::cout << "v_row_stride: " << params.v_row_stride << std::endl;
  std::cout << "q_head_stride: " << params.q_head_stride << std::endl;
  std::cout << "k_head_stride: " << params.k_head_stride << std::endl;
  std::cout << "v_head_stride: " << params.v_head_stride << std::endl;
  std::cout << "h: " << params.h << std::endl;
  std::cout << "h_k: " << params.h_k << std::endl;
  std::cout << "h_h_k_ratio: " << params.h_h_k_ratio << std::endl;

  std::cout << "o_batch_stride: " << params.o_batch_stride << std::endl;
  std::cout << "o_row_stride: " << params.o_row_stride << std::endl;
  std::cout << "o_head_stride: " << params.o_head_stride << std::endl;
  std::cout << "b: " << params.b << std::endl;
  std::cout << "seqlen_q: " << params.seqlen_q << std::endl;
  std::cout << "seqlen_k: " << params.seqlen_k << std::endl;
  std::cout << "d: " << params.d << std::endl;
  std::cout << "seqlen_q_rounded: " << params.seqlen_q_rounded << std::endl;
  std::cout << "seqlen_k_rounded: " << params.seqlen_k_rounded << std::endl;
  std::cout << "d_rounded: " << params.d_rounded << std::endl;
  std::cout << "scale_softmax: " << params.scale_softmax << std::endl;
  std::cout << "scale_softmax_log2: " << params.scale_softmax_log2 << std::endl;
  std::cout << "p_dropout: " << params.p_dropout << std::endl;
  std::cout << "p_dropout_in_uint8_t: " << params.p_dropout_in_uint8_t
            << std::endl;
  std::cout << "rp_dropout: " << params.rp_dropout << std::endl;
  std::cout << "scale_softmax_rp_dropout: " << params.scale_softmax_rp_dropout
            << std::endl;
  std::cout << "is_bf16: " << params.is_bf16 << std::endl;
  std::cout << "is_causal: " << params.is_causal << std::endl;

  std::cout << "do_batch_stride: " << params.do_batch_stride << std::endl;
  std::cout << "do_row_stride: " << params.do_row_stride << std::endl;
  std::cout << "do_head_stride: " << params.do_head_stride << std::endl;
  std::cout << "dq_batch_stride: " << params.dq_batch_stride << std::endl;
  std::cout << "dk_batch_stride: " << params.dk_batch_stride << std::endl;
  std::cout << "dv_batch_stride: " << params.dv_batch_stride << std::endl;
  std::cout << "dq_row_stride: " << params.dq_row_stride << std::endl;
  std::cout << "dk_row_stride: " << params.dk_row_stride << std::endl;
  std::cout << "dv_row_stride: " << params.dv_row_stride << std::endl;
  std::cout << "dq_head_stride: " << params.dq_head_stride << std::endl;
  std::cout << "dk_head_stride: " << params.dk_head_stride << std::endl;
  std::cout << "dv_head_stride: " << params.dv_head_stride << std::endl;
}

// Find the number of splits that maximizes the occupancy. For example, if we
// have batch * n_heads = 48 and we have 108 SMs, having 2 splits (efficiency =
// 0.89) is better than having 3 splits (efficiency = 0.67). However, we also
// don't want too many splits as that would incur more HBM reads/writes. So we
// find the best efficiency, then find the smallest number of splits that gets
// 85% of the best efficiency.
inline int num_splits_heuristic(int batch_nheads_mblocks, int num_SMs,
                                int num_n_blocks, int max_splits) {
  // If we have enough to almost fill the SMs, then just use 1 split
  if (batch_nheads_mblocks >= 0.8f * num_SMs) {
    return 1;
  }
  max_splits = std::min({max_splits, num_SMs, num_n_blocks});
  float max_efficiency = 0.f;
  std::vector<float> efficiency;
  efficiency.reserve(max_splits);
  auto ceildiv = [](int a, int b) { return (a + b - 1) / b; };
  // Some splits are not eligible. For example, if we have 64 blocks and choose
  // 11 splits, we'll have 6 * 10 + 4 blocks. If we choose 12 splits, we'll have
  // 6 * 11 + (-2) blocks (i.e. it's 11 splits anyway). So we check if the
  // number of blocks per split is the same as the previous num_splits.
  auto is_split_eligible = [&ceildiv, &num_n_blocks](int num_splits) {
    return num_splits == 1 || ceildiv(num_n_blocks, num_splits) !=
                                  ceildiv(num_n_blocks, num_splits - 1);
  };
  for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
    if (!is_split_eligible(num_splits)) {
      efficiency.push_back(0.f);
    } else {
      float n_waves = float(batch_nheads_mblocks * num_splits) / num_SMs;
      float eff = n_waves / ceil(n_waves);
      // printf("num_splits = %d, eff = %f\n", num_splits, eff);
      if (eff > max_efficiency) {
        max_efficiency = eff;
      }
      efficiency.push_back(eff);
    }
  }
  for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
    if (!is_split_eligible(num_splits)) {
      continue;
    }
    if (efficiency[num_splits - 1] >= 0.85 * max_efficiency) {
      // printf("num_splits chosen = %d\n", num_splits);
      return num_splits;
    }
  }
  return 1;
}

void run_fwd(Flash_fwd_params params, hipStream_t stream) {
  auto head_dim = params.d;

  if (head_dim <= 32) {
    run_mha_fwd_<cutlass::half_t, 32>(params, stream);
  } else if (head_dim <= 64) {
    run_mha_fwd_<cutlass::half_t, 64>(params, stream);
  } else if (head_dim <= 96) {
    run_mha_fwd_<cutlass::half_t, 96>(params, stream);
  } else  if (head_dim <= 128) {
    run_mha_fwd_<cutlass::half_t, 128>(params, stream);
  } else  if (head_dim <= 160) {
    run_mha_fwd_<cutlass::half_t, 160>(params, stream);
  } else  if (head_dim <= 192) {
    run_mha_fwd_<cutlass::half_t, 192>(params, stream);
  } else  if (head_dim <= 224) {
    run_mha_fwd_<cutlass::half_t, 224>(params, stream);
  } else {
    run_mha_fwd_<cutlass::half_t, 256>(params, stream);
  }
}

void run_fwd_kvcache(Flash_fwd_params params, hipStream_t stream) {
  auto head_dim = params.d;

  if (head_dim <= 32) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 32>(params, stream);
  } else if (head_dim <= 64) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 64>(params, stream);
  } else if (head_dim <= 96) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 96>(params, stream);
  } else  if (head_dim <= 128) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 128>(params, stream);
  } else  if (head_dim <= 160) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 160>(params, stream);
  } else  if (head_dim <= 192) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 192>(params, stream);
  } else  if (head_dim <= 224) {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 224>(params, stream);
  } else {
    run_mha_fwd_splitkv_dispatch<cutlass::half_t, 256>(params, stream);
  }
}

void run_bwd(Flash_bwd_params params, hipStream_t stream) {
  auto head_dim = params.d;

  if (head_dim <= 32) {
    run_mha_bwd_<cutlass::half_t, 32>(params, stream);
  } else if (head_dim <= 64) {
    run_mha_bwd_<cutlass::half_t, 64>(params, stream);
  } else if (head_dim <= 96) {
    run_mha_bwd_<cutlass::half_t, 96>(params, stream);
  } else  if (head_dim <= 128) {
    run_mha_bwd_<cutlass::half_t, 128>(params, stream);
  } else  if (head_dim <= 160) {
    run_mha_bwd_<cutlass::half_t, 160>(params, stream);
  } else  if (head_dim <= 192) {
    run_mha_bwd_<cutlass::half_t, 192>(params, stream);
  } else  if (head_dim <= 224) {
    run_mha_bwd_<cutlass::half_t, 224>(params, stream);
  } else {
    run_mha_bwd_<cutlass::half_t, 256>(params, stream);
  }
}

void run_mha(void *q_ptr, void *k_ptr, void *v_ptr, void *o_ptr,
             void *softmax_lse_ptr, void *softmax_ptr, void *rng_state_ptr,

             uint32_t q_batch_stride, uint32_t k_batch_stride,
             uint32_t v_batch_stride, uint32_t o_batch_stride,

             uint32_t q_row_stride, uint32_t k_row_stride,
             uint32_t v_row_stride, uint32_t o_row_stride,

             uint32_t q_head_stride, uint32_t k_head_stride,
             uint32_t v_head_stride, uint32_t o_head_stride,

             uint32_t b, uint32_t h, uint32_t h_k, uint32_t d,
             uint32_t d_rounded, float softmax_scale,

             uint32_t seqlen_q, uint32_t seqlen_k, uint32_t seqlen_q_rounded,
             uint32_t seqlen_k_rounded,

             float p_dropout, int window_size_left, int window_size_right,
             hipStream_t stream) {
  Flash_fwd_params params;
  // Reset the parameters
  memset(&params, 0, sizeof(params));

  // Set the pointers and strides.
  params.q_ptr = q_ptr;
  params.k_ptr = k_ptr;
  params.v_ptr = v_ptr;
  params.o_ptr = o_ptr;

  params.softmax_lse_ptr = softmax_lse_ptr;

  // All stride are in elements, not bytes.
  params.q_batch_stride = q_batch_stride;
  params.k_batch_stride = k_batch_stride;
  params.v_batch_stride = v_batch_stride;
  params.o_batch_stride = o_batch_stride;

  params.q_row_stride = q_row_stride;
  params.k_row_stride = k_row_stride;
  params.v_row_stride = v_row_stride;
  params.o_row_stride = o_row_stride;
  params.q_head_stride = q_head_stride;
  params.k_head_stride = k_head_stride;
  params.v_head_stride = v_head_stride;
  params.o_head_stride = o_head_stride;

  // Set the dimensions.
  params.b = b;
  params.h = h;
  params.h_k = h_k;
  params.h_h_k_ratio = h / h_k;
  params.seqlen_q = seqlen_q;
  params.seqlen_k = seqlen_k;
  params.seqlen_q_rounded = seqlen_q_rounded;
  params.seqlen_k_rounded = seqlen_k_rounded;
  params.d = d;
  params.d_rounded = d_rounded;

  // Set the different scale values.
  params.scale_softmax = softmax_scale;
  params.scale_softmax_log2 = softmax_scale * M_LOG2E;

  params.p_dropout = 1.f - p_dropout; // probability to keep
  params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
  params.rp_dropout = 1.f / params.p_dropout;
  params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
  params.is_bf16 = 0;
  params.cu_seqlens_q = nullptr;
  params.cu_seqlens_k = nullptr;
  params.seqused_k = nullptr;
  params.p_ptr = softmax_ptr; // used for `return_softmax`.
  params.rng_state = static_cast<uint64_t *>(rng_state_ptr);
  params.is_causal = window_size_left < 0 && window_size_right == 0;

  if (window_size_left < 0 && window_size_right >= 0) {
    window_size_left = seqlen_k;
  }
  if (window_size_left >= 0 && window_size_right < 0) {
    window_size_right = seqlen_k;
  }
  params.window_size_left = window_size_left;
  params.window_size_right = window_size_right;
  params.is_seqlens_k_cumulative = true;
  params.alibi_slopes_ptr = nullptr;
  // print_Flash_fwd_params(params);

  run_fwd(params, stream);
}

void run_mha_bwd(void *q_ptr, void *k_ptr, void *v_ptr, void *o_ptr,
                 void *dout_ptr, void *dq_ptr, void *dk_ptr, void *dv_ptr,
                 void *dq_accum_ptr,
                 void *softmax_lse_ptr, void *dsoftmax_sum_ptr,
                 void *rng_state_ptr,

                 uint32_t q_batch_stride, uint32_t k_batch_stride,
                 uint32_t v_batch_stride, uint32_t o_batch_stride,

                 uint32_t q_row_stride, uint32_t k_row_stride,
                 uint32_t v_row_stride, uint32_t o_row_stride,

                 uint32_t q_head_stride, uint32_t k_head_stride,
                 uint32_t v_head_stride, uint32_t o_head_stride,

                 uint32_t b, uint32_t h, uint32_t h_k, uint32_t d,
                 uint32_t d_rounded, float softmax_scale,

                 uint32_t seqlen_q, uint32_t seqlen_k,
                 uint32_t seqlen_q_rounded, uint32_t seqlen_k_rounded,

                 float p_dropout, int window_size_left, int window_size_right,
                 hipStream_t stream) {
  Flash_bwd_params params;
  // Reset the parameters
  memset(&params, 0, sizeof(params));

  // Set the pointers and strides.
  params.q_ptr = q_ptr;
  params.k_ptr = k_ptr;
  params.v_ptr = v_ptr;
  params.o_ptr = o_ptr;

  params.dq_ptr = dq_ptr;
  params.dk_ptr = dk_ptr;
  params.dv_ptr = dv_ptr;
  params.do_ptr = dout_ptr;

  params.dq_accum_ptr = dq_accum_ptr;
  params.dk_accum_ptr = nullptr;
  params.dv_accum_ptr = nullptr;

  params.softmax_lse_ptr = softmax_lse_ptr;

  // All stride are in elements, not bytes.
  params.q_batch_stride = q_batch_stride;
  params.k_batch_stride = k_batch_stride;
  params.v_batch_stride = v_batch_stride;
  params.o_batch_stride = o_batch_stride;

  params.q_row_stride = q_row_stride;
  params.k_row_stride = k_row_stride;
  params.v_row_stride = v_row_stride;
  params.o_row_stride = o_row_stride;
  params.q_head_stride = q_head_stride;
  params.k_head_stride = k_head_stride;
  params.v_head_stride = v_head_stride;
  params.o_head_stride = o_head_stride;

  params.dq_batch_stride = q_batch_stride;
  params.dk_batch_stride = k_batch_stride;
  params.dv_batch_stride = v_batch_stride;
  params.do_batch_stride = o_batch_stride;

  params.dq_row_stride = q_row_stride;
  params.dk_row_stride = k_row_stride;
  params.dv_row_stride = v_row_stride;
  params.do_row_stride = o_row_stride;
  params.dq_head_stride = q_head_stride;
  params.dk_head_stride = k_head_stride;
  params.dv_head_stride = v_head_stride;
  params.do_head_stride = o_head_stride;

  // Set the dimensions.
  params.b = b;
  params.h = h;
  params.h_k = h_k;
  params.h_h_k_ratio = h / h_k;
  params.seqlen_q = seqlen_q;
  params.seqlen_k = seqlen_k;
  params.seqlen_q_rounded = seqlen_q_rounded;
  params.seqlen_k_rounded = seqlen_k_rounded;
  params.d = d;
  params.d_rounded = d_rounded;
  params.is_causal = window_size_left < 0 && window_size_right == 0;
  if (window_size_left < 0 && window_size_right >= 0) {
    window_size_left = seqlen_k;
  }
  if (window_size_left >= 0 && window_size_right < 0) {
    window_size_right = seqlen_k;
  }
  params.window_size_left = window_size_left;
  params.window_size_right = window_size_right;
  params.is_seqlens_k_cumulative = true;

  // Set the different scale values.
  params.scale_softmax = softmax_scale;
  params.scale_softmax_log2 = softmax_scale * M_LOG2E;

  params.p_dropout = 1.f - p_dropout; // probability to keep
  params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
  params.rp_dropout = 1.f / params.p_dropout;
  params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
  params.is_bf16 = 0;
  params.cu_seqlens_q = nullptr;
  params.cu_seqlens_k = nullptr;
  params.seqused_k = nullptr;
  params.p_ptr = nullptr; // used for `return_softmax`, no use in bwd
  params.dsoftmax_sum = dsoftmax_sum_ptr;
  params.rng_state = static_cast<uint64_t *>(rng_state_ptr);
  params.alibi_slopes_ptr = nullptr;
  // print_Flash_bwd_params(params);

  run_bwd(params, stream);
}

void run_mha_fwd_with_kvcache(
    void *q_ptr, void *k_ptr, void *v_ptr, void *knew_ptr, void *vnew_ptr,
    void *seqlens_k_, void *o_ptr, void *softmax_lse_ptr,

    uint32_t q_batch_stride, uint32_t k_batch_stride, uint32_t v_batch_stride,
    uint32_t knew_batch_stride, uint32_t vnew_batch_stride,
    uint32_t o_batch_stride,

    uint32_t q_row_stride, uint32_t k_row_stride, uint32_t v_row_stride,
    uint32_t knew_row_stride, uint32_t vnew_row_stride, uint32_t o_row_stride,

    uint32_t q_head_stride, uint32_t k_head_stride, uint32_t v_head_stride,
    uint32_t knew_head_stride, uint32_t vnew_head_stride,
    uint32_t o_head_stride,

    uint32_t b, uint32_t h, uint32_t h_k, uint32_t d, uint32_t d_rounded,
    uint32_t seqlen_knew, float softmax_scale,

    uint32_t seqlen_q, uint32_t seqlen_k, uint32_t seqlen_q_rounded,
    uint32_t seqlen_k_rounded,

    int window_size_left, int window_size_right, hipStream_t stream) {
  Flash_fwd_params params;
  // Reset the parameters
  memset(&params, 0, sizeof(params));

  // Set the pointers and strides.
  params.q_ptr = q_ptr;
  params.k_ptr = k_ptr;
  params.v_ptr = v_ptr;
  params.o_ptr = o_ptr;

  params.softmax_lse_ptr = softmax_lse_ptr;

  // All stride are in elements, not bytes.
  params.q_batch_stride = q_batch_stride;
  params.k_batch_stride = k_batch_stride;
  params.v_batch_stride = v_batch_stride;
  params.o_batch_stride = o_batch_stride;

  params.q_row_stride = q_row_stride;
  params.k_row_stride = k_row_stride;
  params.v_row_stride = v_row_stride;
  params.o_row_stride = o_row_stride;
  params.q_head_stride = q_head_stride;
  params.k_head_stride = k_head_stride;
  params.v_head_stride = v_head_stride;
  params.o_head_stride = o_head_stride;

  // Set the dimensions.
  params.b = b;
  params.h = h;
  params.h_k = h_k;
  params.h_h_k_ratio = h / h_k;
  params.seqlen_q = seqlen_q;
  params.seqlen_k = seqlen_k;
  params.seqlen_q_rounded = seqlen_q_rounded;
  params.seqlen_k_rounded = seqlen_k_rounded;
  params.d = d;
  params.d_rounded = d_rounded;

  // Set the different scale values.
  params.scale_softmax = softmax_scale;
  params.scale_softmax_log2 = softmax_scale * M_LOG2E;

  params.p_dropout = 1.f; // probability to keep
  params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
  params.rp_dropout = 1.f / params.p_dropout;
  params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;

  params.is_bf16 = 0;
  params.cu_seqlens_q = nullptr;
  params.cu_seqlens_k = static_cast<int *>(seqlens_k_);
  params.seqused_k = nullptr;
  params.p_ptr = nullptr; // used for `return_softmax`.
  params.rng_state = nullptr;
  params.alibi_slopes_ptr = nullptr;
  params.page_block_size = 1;
  params.is_causal = window_size_left < 0 && window_size_right == 0;

  if (window_size_left < 0 && window_size_right >= 0) {
    window_size_left = seqlen_k;
  }
  if (window_size_left >= 0 && window_size_right < 0) {
    window_size_right = seqlen_k;
  }
  params.window_size_left = window_size_left;
  params.window_size_right = window_size_right;
  params.is_seqlens_k_cumulative = false;

  params.seqlen_knew = seqlen_knew;
  params.knew_ptr = knew_ptr;
  params.vnew_ptr = vnew_ptr;
  // All stride are in elements, not bytes.
  params.knew_batch_stride = knew_batch_stride;
  params.vnew_batch_stride = vnew_batch_stride;
  params.knew_row_stride = knew_row_stride;
  params.vnew_row_stride = vnew_row_stride;
  params.knew_head_stride = knew_head_stride;
  params.vnew_head_stride = vnew_head_stride;

  // TODO: ROPE support TBD
  params.rotary_dim = 0;

  // This needs to match with run_mha_fwd_splitkv_dispatch
  // const int head_size = round_multiple(head_size_og, 8);
  const int block_n = h <= 64 ? 256 : (h <= 128 ? 128 : 64);
  const int num_n_blocks = (seqlen_k + block_n - 1) / block_n;
  // Technically kBlockM = 64 only for the splitKV kernels, not the standard
  // kernel. In any case we don't expect seqlen_q to be larger than 64 for
  // inference.
  const int num_m_blocks = (seqlen_q + 64 - 1) / 64;
  // hipDeviceProp_t dprops;
  // hipGetDeviceProperties(&dprops, 0);
  // params.num_splits = num_splits_heuristic(
  //     b * h_k * num_m_blocks, dprops->multiProcessorCount, num_n_blocks,
  //     128);
  // static_assert(params.num_splits <= 128 && "num_splits > 128 not
  // supported");
  params.num_splits = 1;
  // TODO: support > 1 split
  // if (params.num_splits > 1) {
  //   at::Tensor softmax_lse_accum =
  //       torch::empty({params.num_splits, batch_size, num_heads, seqlen_q},
  //                    opts.dtype(at::kFloat));
  //   at::Tensor out_accum = torch::empty(
  //       {params.num_splits, batch_size, num_heads, seqlen_q,
  //       head_size_rounded}, opts.dtype(at::kFloat));
  //   params.softmax_lseaccum_ptr = softmax_lse_accum.data_ptr();
  //   params.oaccum_ptr = out_accum.data_ptr();
  // }

  run_fwd_kvcache(params, stream);
}

int64_t getIntFromVoidPtr(void *data, size_t &pos) {
  int64_t *intPtr =
      reinterpret_cast<int64_t *>(static_cast<char *>(data) + pos);
  pos += sizeof(int64_t);
  return *intPtr;
}

float getFloatFromVoidPtr(void *data, size_t &pos) {
  float *floatPtr = reinterpret_cast<float *>(static_cast<char *>(data) + pos);
  pos += sizeof(float);
  return *floatPtr;
}

#ifdef __cplusplus
extern "C" {
#endif

void run_flash_attn_fwd(void **tensors, void *extra_args, hipStream_t stream) {
  size_t pos = 0;
  auto q_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto q_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto q_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto b = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto h = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto h_k = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto d = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto d_rounded = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto softmax_scale = static_cast<float>(getFloatFromVoidPtr(extra_args, pos));
  auto seqlen_q = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_k = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_q_rounded =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_k_rounded =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto p_dropout = static_cast<float>(getFloatFromVoidPtr(extra_args, pos));
  auto window_size_left =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto window_size_right =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));

  // tensors: q, k, v, rng_state, o, softmax_lse, softmax_sum
  run_mha(tensors[0], tensors[1], tensors[2], tensors[4], tensors[5],
          tensors[6], tensors[3],
          /*q_batch_stride*/ q_batch_stride,
          /*k_batch_stride*/ k_batch_stride,
          /*v_batch_stride*/ v_batch_stride,
          /*o_batch_stride*/ o_batch_stride,
          /*q_row_stride*/ q_row_stride,
          /*k_row_stride*/ k_row_stride,
          /*v_row_stride*/ v_row_stride,
          /*o_row_stride*/ o_row_stride,
          /*q_head_stride*/ q_head_stride,
          /*k_head_stride*/ k_head_stride,
          /*v_head_stride*/ v_head_stride,
          /*o_head_stride*/ o_head_stride,
          /*b*/ b,
          /*h*/ h,
          /*h_k*/ h_k,
          /*d*/ d,
          /*d_rounded*/ d_rounded,
          /*softmax_scale*/ softmax_scale,
          /*seqlen_q*/ seqlen_q,
          /*seqlen_k*/ seqlen_k,
          /*seqlen_q_rounded*/ seqlen_q_rounded,
          /*seqlen_k_rounded*/ seqlen_k_rounded,
          /*p_dropout*/ p_dropout,
          /*window_size_left*/ window_size_left,
          /*window_size_right*/ window_size_right, stream);
}

void run_flash_attn_bwd(void **tensors, void *extra_args, hipStream_t stream) {
  size_t pos = 0;
  auto q_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto q_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto q_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto b = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto h = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto h_k = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto d = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto d_rounded = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto softmax_scale = static_cast<float>(getFloatFromVoidPtr(extra_args, pos));
  auto seqlen_q = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_k = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_q_rounded =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_k_rounded =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto p_dropout = static_cast<float>(getFloatFromVoidPtr(extra_args, pos));
  auto window_size_left =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto window_size_right =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));

  //tensors: dout, q, k, v, out, softmax_lse, rng_state, dq, dk, dv, d_softmax, dq_accum
  run_mha_bwd(tensors[1], tensors[2], tensors[3], tensors[4], tensors[0],
              tensors[7], tensors[8], tensors[9], tensors[11], tensors[5],
              tensors[10], tensors[6],
              /*q_batch_stride*/ q_batch_stride,
              /*k_batch_stride*/ k_batch_stride,
              /*v_batch_stride*/ v_batch_stride,
              /*o_batch_stride*/ o_batch_stride,
              /*q_row_stride*/ q_row_stride,
              /*k_row_stride*/ k_row_stride,
              /*v_row_stride*/ v_row_stride,
              /*o_row_stride*/ o_row_stride,
              /*q_head_stride*/ q_head_stride,
              /*k_head_stride*/ k_head_stride,
              /*v_head_stride*/ v_head_stride,
              /*o_head_stride*/ o_head_stride,
              /*b*/ b,
              /*h*/ h,
              /*h_k*/ h_k,
              /*d*/ d,
              /*d_rounded*/ d_rounded,
              /*softmax_scale*/ softmax_scale,
              /*seqlen_q*/ seqlen_q,
              /*seqlen_k*/ seqlen_k,
              /*seqlen_q_rounded*/ seqlen_q_rounded,
              /*seqlen_k_rounded*/ seqlen_k_rounded,
              /*p_dropout*/ p_dropout,
              /*window_size_left*/ window_size_left,
              /*window_size_right*/ window_size_right, stream);
}

void run_flash_attn_kvcache(void **tensors, void *extra_args,
                            hipStream_t stream) {
  size_t pos = 0;
  auto q_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto knew_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto vnew_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_batch_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto q_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto knew_row_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto vnew_row_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_row_stride = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto q_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto k_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto v_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto knew_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto vnew_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto o_head_stride =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto b = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto h = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto h_k = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto d = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto d_rounded = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_knew = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto softmax_scale = static_cast<float>(getFloatFromVoidPtr(extra_args, pos));
  auto seqlen_q = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_k = static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_q_rounded =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto seqlen_k_rounded =
      static_cast<uint32_t>(getIntFromVoidPtr(extra_args, pos));
  auto window_size_left = static_cast<int>(getIntFromVoidPtr(extra_args, pos));
  auto window_size_right = static_cast<int>(getIntFromVoidPtr(extra_args, pos));

  run_mha_fwd_with_kvcache(tensors[0], tensors[1], tensors[2], tensors[3],
                           tensors[4], tensors[5], tensors[6], tensors[7],
                           /*q_batch_stride*/ q_batch_stride,
                           /*k_batch_stride*/ k_batch_stride,
                           /*v_batch_stride*/ v_batch_stride,
                           /*knew_batch_stride*/ knew_batch_stride,
                           /*vnew_batch_stride*/ vnew_batch_stride,
                           /*o_batch_stride*/ o_batch_stride,
                           /*q_row_stride*/ q_row_stride,
                           /*k_row_stride*/ k_row_stride,
                           /*v_row_stride*/ v_row_stride,
                           /*knew_row_stride*/ knew_row_stride,
                           /*vnew_row_stride*/ vnew_row_stride,
                           /*o_row_stride*/ o_row_stride,
                           /*q_head_stride*/ q_head_stride,
                           /*k_head_stride*/ k_head_stride,
                           /*v_head_stride*/ v_head_stride,
                           /*knew_head_stride*/ knew_head_stride,
                           /*vnew_head_stride*/ vnew_head_stride,
                           /*o_head_stride*/ o_head_stride,
                           /*b*/ b,
                           /*h*/ h,
                           /*h_k*/ h_k,
                           /*d*/ d,
                           /*d_rounded*/ d_rounded,
                           /*seqlen_knew*/ seqlen_knew,
                           /*softmax_scale*/ softmax_scale,
                           /*seqlen_q*/ seqlen_q,
                           /*seqlen_k*/ seqlen_k,
                           /*seqlen_q_rounded*/ seqlen_q_rounded,
                           /*seqlen_k_rounded*/ seqlen_k_rounded,
                           /*window_size_left*/ window_size_left,
                           /*window_size_right*/ window_size_right, stream);
}
#ifdef __cplusplus
}
#endif
