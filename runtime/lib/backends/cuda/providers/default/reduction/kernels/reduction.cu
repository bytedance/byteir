#include "hip/hip_runtime.h"
// Copyright (c) Megvii Inc.
// Licensed under the Apache License.
// ===========================================================================
// Modification Copyright 2022 ByteDance Ltd. and/or its affiliates.

#include "./reduction.h"
#include "./reduction_helper.h"
#include <algorithm>
#include <cstdint>

#define DIVUP(x, y) (((x) + (y)-1) / (y))

namespace brt {
namespace cuda {
namespace kernel {
namespace reduction {
/*!
 * each block has (1 << block_size_log2) threads and process fixed number of
 * rows; each row is processed by (1 << nr_thread_per_row_log2) threads.
 *
 * need a padding of max_nr_threads_per_row/2 elements after shared memory
 */
template <int block_size_log2, int max_nr_threads_per_row, class Op,
          int warp_size>
__global__ void kern_column(Op op, uint32_t A, uint32_t B,
                            uint32_t nr_thread_per_row_log2,
                            uint32_t sm_width_byte) {
  typedef typename Op::wtype wtype;
  // shared mem: matrix(nr_row_per_block, nr_thread_per_row)
  extern __shared__ uint8_t sub_block_raw[];

  uint32_t nr_row_per_block = 1 << (block_size_log2 - nr_thread_per_row_log2),
           nr_thread_per_row = 1 << nr_thread_per_row_log2,
           row_num = threadIdx.x >> nr_thread_per_row_log2,
           // tid in current row
      tid = threadIdx.x - (row_num << nr_thread_per_row_log2),
           a = blockIdx.x * nr_row_per_block + row_num;

  volatile wtype *row = (wtype *)(sub_block_raw + row_num * sm_width_byte);
  // sum columns of src[a0:a1] and store in row
  {
    uint32_t base = min(a, A - 1) * B;
    wtype csum = op.read(base + tid);
    for (int c = tid + nr_thread_per_row; c < B; c += nr_thread_per_row) {
      csum = Op::apply(csum, op.read(base + c));
    }
    row[tid] = csum;
  }

#pragma unroll
  for (uint32_t i = max_nr_threads_per_row / 2; i; i >>= 1) {
    bool cond = nr_thread_per_row >= i * 2 && tid < i;
    if (i >= warp_size) {
      __syncthreads();
    } else {
      /**
       * \warning Since CUDA 9.0, for Volta and Turing architecture,
       * applications that assume reads and writes are implicitly visible
       * to other threads in same warp need to insert the new __syncwarp()
       * warp-wide barrier synchronization instruction between steps where
       * data is exchanged between threads via global or shared memory.
       * For details, please refer to
       * https://docs.nvidia.com/cuda/volta-tuning-guide/index.html
       */
      __syncwarp(0xffffffff);
    }
    if (cond) {
      wtype v0 = row[tid];
      wtype v1 = Op::apply(v0, row[tid + i]);
      row[tid] = v1;
    }
  }

  if (a < A && !tid) {
    op.write(a, row[0]);
  }
}

template <class Op, uint32_t max_nr_threads_per_row, uint32_t block_size_log2,
          uint32_t warp_size>
void _do_run_column(uint32_t A, uint32_t B, hipStream_t stream, const Op &op) {
  typedef typename Op::wtype wtype;
  const uint32_t block_size = 1 << block_size_log2;
  uint32_t nr_thread_per_row = 1, nr_thread_per_row_log2 = 0;

  while (nr_thread_per_row < max_nr_threads_per_row &&
         nr_thread_per_row * 2 <= B) {
    ++nr_thread_per_row_log2;
    nr_thread_per_row *= 2;
  }
  // now: nr_thread_per_row <= B < nr_thread_per_row * 2

  if (B <= max_nr_threads_per_row * 4) {
    // find nr_thread_per_row with minimal wasted threads
    uint32_t min_cost = std::numeric_limits<uint32_t>::max(), min_cost_th = 0;
    for (uint32_t i = warp_size; i <= nr_thread_per_row; i *= 2) {
      uint32_t cost = (i - B % i) % i;
      if (cost < min_cost) {
        min_cost = cost;
        min_cost_th = i;
      }
    }
    if (min_cost_th) {
      nr_thread_per_row = min_cost_th;
      while ((1u << nr_thread_per_row_log2) != nr_thread_per_row)
        --nr_thread_per_row_log2;
    }
  }

  uint32_t nr_row_per_block = block_size / nr_thread_per_row,
           nr_blk = DIVUP(A, nr_row_per_block),
           sm_width_word32 = DIVUP(nr_thread_per_row * sizeof(wtype), 4ul);

  // gcd(sm_width_word32, BANKS) should be 1 to avoid bank confliction
  // iff sm_width_word32 is odd
  sm_width_word32 += !(sm_width_word32 % 2);
  uint32_t sm_width_byte = sm_width_word32 * 4,
           sm_size = nr_row_per_block * sm_width_byte +
                     sizeof(wtype) * max_nr_threads_per_row / 2;

  void (*kptr)(Op op, uint32_t A, uint32_t B, uint32_t nr_thread_per_row_log2,
               uint32_t sm_width_byte);
  if (nr_thread_per_row <= max_nr_threads_per_row / 4) {
    kptr =
        kern_column<block_size_log2, max_nr_threads_per_row / 4, Op, warp_size>;
  } else if (nr_thread_per_row <= max_nr_threads_per_row / 2) {
    kptr =
        kern_column<block_size_log2, max_nr_threads_per_row / 2, Op, warp_size>;
  } else {
    kptr = kern_column<block_size_log2, max_nr_threads_per_row, Op, warp_size>;
  }
  kptr<<<nr_blk, block_size, sm_size, stream>>>(
      op, A, B, nr_thread_per_row_log2, sm_width_byte);
}

// use struct to allow default template arguments in C++-03
/*!
 * \brief start the cuda kernel to reduce in column direction of a matrix
 * \tparam max_nr_threads_per_row max number of threads to reduce each row
 * \tparam block_size_log2 log2 of threads in a block
 * \tparam warp_size size of warp on the device
 */
template <class Op, uint32_t max_nr_threads_per_row = 64,
          uint32_t block_size_log2 = 7, uint32_t warp_size = 32>
struct run_column {
  static void run(uint32_t A, uint32_t B, hipStream_t stream, const Op &op) {
    return _do_run_column<Op, max_nr_threads_per_row, block_size_log2,
                          warp_size>(A, B, stream, op);
  }
};

struct ExecPolicy {
  // (BY, BX) is the blockDim to launch reduce kernel
  ExecPolicy(size_t A, size_t B, size_t C) : A(A), B(B), C(C) {
    // use C to determine BX
    BX = 1;
    while (BX < 32 && BX < C)
      BX *= 2;
    BY = 512 / BX;
    NA = A;
    factor = BY * 4;
    NB = DIVUP(B, factor);
    NC = DIVUP(C, BX);
    {
      nr_reduces = 0;
      size_t tmp = B;
      while (tmp > 1) {
        tmp = DIVUP(tmp, factor);
        ++nr_reduces;
      }
      if (nr_reduces == 0)
        nr_reduces = 1;
    }
  }
  ExecPolicy next() const { return ExecPolicy(A, DIVUP(B, factor), C); }
  size_t factor;
  size_t nr_reduces;
  size_t BY, BX;
  size_t NA, NB, NC;
  size_t A, B, C;
};

// Whenever blockIdx is referenced, bidy_offset and bidz_offset should be added.
// This mechanism is to solve thread block size limitation issue by calling
// multiple kernels from host code.
template <class Operator, class Reader, class Writer, typename wtype,
          uint32_t BX, uint32_t BY, bool sync_within_warp>
__global__ void kern_largeBC(Operator opr, Reader rdr, Writer wtr, uint32_t A,
                             uint32_t B, uint32_t B2, uint32_t C,
                             uint32_t bidy_offset, uint32_t bidz_offset) {
  volatile __shared__ wtype shared[BY][BX];
  wtype s = opr.INIT;
  uint32_t c = threadIdx.x + blockIdx.x * blockDim.x;
  uint32_t a = blockIdx.z + bidz_offset;
  if (c < C) {
    uint32_t base = threadIdx.y + (blockIdx.y + bidy_offset) * 4 * blockDim.y;
    if (base + 0 * blockDim.y < B) {
      s = opr.apply(s, rdr.read(a * B * C + (base + 0 * blockDim.y) * C + c));
    }
    if (base + 1 * blockDim.y < B) {
      s = opr.apply(s, rdr.read(a * B * C + (base + 1 * blockDim.y) * C + c));
    }
    if (base + 2 * blockDim.y < B) {
      s = opr.apply(s, rdr.read(a * B * C + (base + 2 * blockDim.y) * C + c));
    }
    if (base + 3 * blockDim.y < B) {
      s = opr.apply(s, rdr.read(a * B * C + (base + 3 * blockDim.y) * C + c));
    }
  }
  shared[threadIdx.y][threadIdx.x] = s;
  __syncthreads();

  const uint32_t warp_y = 32 / BX;
#pragma unroll
  for (uint32_t k = 256; k > warp_y; k >>= 1) {
    if (BY >= k << 1) {
      if (threadIdx.y < k) {
        shared[threadIdx.y][threadIdx.x] =
            opr.apply(shared[threadIdx.y][threadIdx.x],
                      shared[threadIdx.y + k][threadIdx.x]);
      }
      __syncthreads();
    }
  }
  if (threadIdx.y < warp_y) {
#pragma unroll
    for (uint32_t k = warp_y; k > 0; k >>= 1) {
      if (threadIdx.y < k) {
        shared[threadIdx.y][threadIdx.x] =
            opr.apply(shared[threadIdx.y][threadIdx.x],
                      shared[threadIdx.y + k][threadIdx.x]);
      }
      if (sync_within_warp) {
        __syncthreads();
      }
      /**
       * \warning Since CUDA 9.0, for Volta and Turing architecture,
       * applications that assume reads and writes are implicitly visible
       * to other threads in same warp need to insert the new __syncwarp()
       * warp-wide barrier synchronization instruction between steps where
       * data is exchanged between threads via global or shared memory.
       * For details, please refer to
       * https://docs.nvidia.com/cuda/volta-tuning-guide/index.html
       */
      __syncwarp(0xffffffff);
    }
  }
  if (threadIdx.y == 0 && c < C) {
    uint32_t b2 = blockIdx.y + bidy_offset;
    wtr.write(a * B2 * C + b2 * C + c, shared[0][threadIdx.x]);
  }
}

/**
 * \tparam Operator must have method wtype apply(wtype, wtype)
 * \tparam Operator must have const member INIT
 * \tparam Reader must have method wtype read(size_t idx)
 * \tparam Writer must have method void write(size_t idx, wtype)
 */
template <class Operator, class Reader, class Writer, typename wtype,
          bool sync_within_warp>
void invoke_kernel(const ExecPolicy &p, const Operator &opr, const Reader &rdr,
                   const Writer &wtr, hipStream_t stream) {
  // 32768 thread blocks for each call
#define CHECK(nBX)                                                             \
  if (p.BX == nBX && p.BY == 512 / nBX) {                                      \
    for (size_t bidy_offset = 0; bidy_offset < p.NB; bidy_offset += 32768)     \
      for (size_t bidz_offset = 0; bidz_offset < p.NA; bidz_offset += 32768) { \
        dim3 blocks;                                                           \
        blocks.x = p.NC;                                                       \
        blocks.y = std::min<size_t>(32768, p.NB - bidy_offset);                \
        blocks.z = std::min<size_t>(32768, p.NA - bidz_offset);                \
        kern_largeBC<Operator, Reader, Writer, wtype, nBX, 512 / nBX,          \
                     sync_within_warp>                                         \
            <<<blocks, dim3(p.BX, p.BY), 0, stream>>>(                         \
                opr, rdr, wtr, p.A, p.B, DIVUP(p.B, p.factor), p.C,            \
                bidy_offset, bidz_offset);                                     \
      }                                                                        \
  }
  CHECK(1);
  CHECK(2);
  CHECK(4);
  CHECK(8);
  CHECK(16);
  CHECK(32);
#undef CHECK
}

/**
 * inherit from PublicOperator
 */
template <class PublicOperator> struct PublicReader {
  PublicOperator opr;
  typedef typename PublicOperator::wtype wtype;
  PublicReader(const PublicOperator &opr) : opr(opr) {}
  __device__ wtype read(uint32_t idx) { return opr.read(idx); }
};

/**
 * read from workspace
 */
template <typename wtype> struct WorkspaceReader {
  wtype *workspace;
  WorkspaceReader(wtype *workspace) : workspace(workspace) {}
  __device__ wtype read(uint32_t idx) { return workspace[idx]; }
};

/**
 * inherit from PublicOperator
 */
template <class PublicOperator> struct PublicWriter {
  PublicOperator opr;
  typedef typename PublicOperator::wtype wtype;
  PublicWriter(const PublicOperator &opr) : opr(opr) {}
  __device__ void write(uint32_t idx, wtype value) { opr.write(idx, value); }
};

/**
 * write to workspace
 */
template <typename wtype> struct WorkspaceWriter {
  wtype *workspace;
  WorkspaceWriter(wtype *workspace) : workspace(workspace) {}
  __device__ void write(uint32_t idx, wtype value) { workspace[idx] = value; }
};

/**
 * \tparam PublicOperator
 *      must have typedef for wtype
 *      must have const static member wtype INIT
 *      must have method wtype read(uint32_t idx)
 *      must have method wtype apply(const wtype &, const wtype &)
 *      must have method void write(uint32_t idx, const wtype &)
 */
template <class PublicOperator, bool sync_within_warp>
void run_largeBC(typename PublicOperator::wtype *workspace, size_t A, size_t B,
                 size_t C, hipStream_t stream, const PublicOperator &opr) {
  typedef typename PublicOperator::wtype wtype;
  ExecPolicy p(A, B, C);
  if (p.nr_reduces == 1) {
    PublicReader<PublicOperator> rdr(opr);
    PublicWriter<PublicOperator> wtr(opr);
    invoke_kernel<PublicOperator, PublicReader<PublicOperator>,
                  PublicWriter<PublicOperator>, wtype, sync_within_warp>(
        p, opr, rdr, wtr, stream);
  } else if (p.nr_reduces == 2) {
    PublicReader<PublicOperator> rdr1(opr);
    WorkspaceWriter<wtype> wtr1(workspace);
    WorkspaceReader<wtype> rdr2(workspace);
    PublicWriter<PublicOperator> wtr2(opr);
    invoke_kernel<PublicOperator, PublicReader<PublicOperator>,
                  WorkspaceWriter<wtype>, wtype, sync_within_warp>(
        p, opr, rdr1, wtr1, stream);
    p = p.next();
    invoke_kernel<PublicOperator, WorkspaceReader<wtype>,
                  PublicWriter<PublicOperator>, wtype, sync_within_warp>(
        p, opr, rdr2, wtr2, stream);
  } else {
    wtype *workspace1 = workspace;
    size_t B2 = DIVUP(B, p.factor);
    wtype *workspace2 = workspace + A * B2 * C;
    size_t nr_reduces = p.nr_reduces;

    {
      PublicReader<PublicOperator> rdr(opr);
      WorkspaceWriter<wtype> wtr(workspace1);
      invoke_kernel<PublicOperator, PublicReader<PublicOperator>,
                    WorkspaceWriter<wtype>, wtype, sync_within_warp>(
          p, opr, rdr, wtr, stream);
    }
    p = p.next();
    wtype *current = workspace1;
    wtype *next = workspace2;
    for (size_t i = 1; i < nr_reduces; ++i) {
      WorkspaceReader<wtype> rdr(current);
      if (i + 1 == nr_reduces) {
        PublicWriter<PublicOperator> wtr(opr);
        invoke_kernel<PublicOperator, WorkspaceReader<wtype>,
                      PublicWriter<PublicOperator>, wtype, sync_within_warp>(
            p, opr, rdr, wtr, stream);
      } else {
        WorkspaceWriter<wtype> wtr(next);
        invoke_kernel<PublicOperator, WorkspaceReader<wtype>,
                      WorkspaceWriter<wtype>, wtype, sync_within_warp>(
            p, opr, rdr, wtr, stream);
      }
      std::swap(next, current);
      p = p.next();
    }
  }
}

template <typename wtype>
size_t get_workspace_largeBC(size_t A, size_t B, size_t C) {
  ExecPolicy p(A, B, C);
  if (p.nr_reduces == 1) {
    // direct reduce
    return 0;
  } else if (p.nr_reduces == 2) {
    // src->workspace->dst
    size_t B2 = DIVUP(B, p.factor);
    return sizeof(wtype) * A * B2 * C;
  } else {
    // src->workspace1->workspace2->dst
    size_t B2 = DIVUP(B, p.factor);
    size_t B3 = DIVUP(B2, p.factor);
    return sizeof(wtype) * A * B2 * C + sizeof(wtype) * A * B3 * C;
  }
}

bool use_reduce_column(size_t A, size_t B, size_t C) {
  return C == 1 && (B <= A * 4 || B <= 32);
}
} // namespace reduction

template <typename T, typename Op>
void call_reduce(const T *input, T *output, size_t A, size_t B, size_t C,
                 void *workspace, hipStream_t stream) {
  Op opr(const_cast<T *>(input), output, B);
  if (reduction::use_reduce_column(A, B, C)) {
    reduction::run_column<Op>::run(A, B, stream, opr);
  } else {
    reduction::run_largeBC<Op, false>(static_cast<float *>(workspace), A, B, C,
                                      stream, opr);
  }
}

template <typename wtype>
size_t get_reduce_workspace_in_bytes(size_t A, size_t B, size_t C) {
  if (reduction::use_reduce_column(A, B, C)) {
    return 0;
  }
  return reduction::get_workspace_largeBC<wtype>(A, B, C);
}

template void call_reduce<__half, reduction::SumOp<__half, __half, float>>(
    const __half *, __half *, size_t, size_t, size_t, void *, hipStream_t);
template void call_reduce<float, reduction::SumOp<float, float, float>>(
    const float *, float *, size_t, size_t, size_t, void *, hipStream_t);
template void call_reduce<float, reduction::MaxOp<float, float, float>>(
    const float *, float *, size_t, size_t, size_t, void *, hipStream_t);
template void call_reduce<__half, reduction::MaxOp<__half, __half, float>>(
    const __half *, __half *, size_t, size_t, size_t, void *, hipStream_t);
template size_t get_reduce_workspace_in_bytes<float>(size_t, size_t, size_t);
template size_t get_reduce_workspace_in_bytes<__half>(size_t, size_t, size_t);

} // namespace kernel
} // namespace cuda
} // namespace brt