//===- cutlass_blas.cu ----------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "cutlass/cutlass.h"
#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wstrict-aliasing"
#pragma GCC diagnostic ignored "-Wsign-compare"
#include "cutlass/gemm/device/gemm_batched.h"
#pragma GCC diagnostic pop
#include "cutlass/layout/matrix.h"

namespace brt {
namespace cuda {
namespace kernel {

// cutlass batch matmul implementation
template <typename T>
cutlass::Status
cutlass_batch_matmul(const T *A, int lda, long long int batch_stride_A,
                     const T *B, int ldb, long long int batch_stride_B, T *C,
                     int ldc, long long int batch_stride_C, int batch_count,
                     int m, int n, int k, T alpha, T beta,
                     hipStream_t stream = nullptr) {
  using Gemm = cutlass::gemm::device::GemmBatched<T, cutlass::layout::RowMajor,
                                                  T, cutlass::layout::RowMajor,
                                                  T, cutlass::layout::RowMajor>;
  Gemm gemm_op;
  return gemm_op({{m, n, k},
                  {A, lda},
                  batch_stride_A,
                  {B, ldb},
                  batch_stride_B,
                  {C, ldc},
                  batch_stride_C,
                  {C, ldc},
                  batch_stride_C,
                  {alpha, beta},
                  batch_count},
                 nullptr, stream);
}

// instantiate
template cutlass::Status
cutlass_batch_matmul<float>(const float *, int, long long int, const float *,
                            int, long long int, float *, int, long long int,
                            int, int, int, int, float, float, hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
