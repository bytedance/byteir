
#include <hip/hip_runtime.h>
//===- elementwise.cu -----------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

namespace brt {
namespace cuda {
namespace kernel {

template <typename T>
__global__ void add_kernel(const T *input_1, const T *input_2, T *output,
                           int n) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < n) {
    output[idx] = input_1[idx] + input_2[idx];
  }
}

// instantiate
template __global__ void add_kernel<float>(const float *, const float *,
                                           float *, int);
template __global__ void add_kernel<int>(const int *, const int *, int *, int);

} // namespace kernel
} // namespace cuda
} // namespace brt
