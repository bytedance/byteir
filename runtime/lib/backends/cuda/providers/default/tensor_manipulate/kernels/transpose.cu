//===- transpose.cu -------------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace brt {
namespace cuda {
namespace kernel {

template <typename T>
__global__ void transpose_naive_2d_kernel(const T *input, T *output, int m,
                                          int n) {
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy < m && ix < n) {
    int in_idx = iy * n + ix;
    int out_idx = ix * m + iy;
    output[out_idx] = input[in_idx];
  }
}

template <typename T>
void transpose_naive_2d(const T *input, T *output, int m, int n, dim3 grid,
                        dim3 block, hipStream_t stream) {
  transpose_naive_2d_kernel<T><<<grid, block, 0, stream>>>(input, output, m, n);
}

// instantiate
template void transpose_naive_2d<float>(const float *, float *, int, int, dim3,
                                        dim3, hipStream_t);
template void transpose_naive_2d<__half>(const __half *, __half *, int, int,
                                         dim3, dim3, hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
