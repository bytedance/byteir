//===- transpose.cu -------------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>

namespace brt {
namespace cuda {
namespace kernel {
constexpr int32_t kMaxGridDim = 65535;
template <typename T>
__global__ void transpose_naive_2d_kernel(const T *input, T *output, int m,
                                          int n) {
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  if (iy < m && ix < n) {
    int in_idx = iy * n + ix;
    int out_idx = ix * m + iy;
    output[out_idx] = input[in_idx];
  }
}

template <typename T>
void transpose_naive_2d(const T *input, T *output, int m, int n, dim3 grid,
                        dim3 block, hipStream_t stream) {
  transpose_naive_2d_kernel<T><<<grid, block, 0, stream>>>(input, output, m, n);
}

template <typename T, int32_t TileSizeX, int32_t TileSizeY, int32_t BlockSize>
__global__ void batch_transpose_kernel(const int32_t total_tile_num,
                                       const int32_t tile_num_in_dim0,
                                       const int32_t tile_num_in_dim1,
                                       const int32_t tile_per_sample,
                                       const int32_t row, const int32_t col,
                                       void *__restrict__ inp_ptr,
                                       void *__restrict__ out_ptr) {
  __shared__ T tile_in_shmem[TileSizeX][TileSizeY];
  for (int32_t i = blockIdx.x, step_tile = gridDim.x; i < total_tile_num;
       i += step_tile) {
    const int32_t batch_idx = i / tile_per_sample;
    const int32_t remainder = i - batch_idx * tile_per_sample;
    const int32_t dim0_idx = remainder / tile_num_in_dim1;
    const int32_t dim1_idx = remainder - dim0_idx * tile_num_in_dim1;

    T *inp_tile_gmem = reinterpret_cast<T *>(inp_ptr);
    T *out_tile_gmem = reinterpret_cast<T *>(out_ptr);
    inp_tile_gmem += batch_idx * row * col + dim0_idx * TileSizeX * col +
                     dim1_idx * TileSizeY;
    out_tile_gmem += batch_idx * row * col + dim1_idx * TileSizeY * row +
                     dim0_idx * TileSizeX;

    int32_t range_0 = dim0_idx < tile_num_in_dim0 - 1
                          ? TileSizeX
                          : row - dim0_idx * TileSizeX;
    int32_t range_1 = dim1_idx < tile_num_in_dim1 - 1
                          ? TileSizeY
                          : col - dim1_idx * TileSizeY;
    constexpr int32_t row_num_per_iter = BlockSize / TileSizeY;
    constexpr int32_t col_num_per_iter = BlockSize / TileSizeX;

    int32_t tile_row_idx = threadIdx.x / TileSizeY;
    int32_t tile_col_idx = threadIdx.x - tile_row_idx * TileSizeY;
    for (int32_t j = tile_row_idx; j < range_0; j += row_num_per_iter) {
      if (tile_col_idx < range_1) {
        tile_in_shmem[j][tile_col_idx ^ j] =
            inp_tile_gmem[j * col + tile_col_idx];
      }
    }
    __syncthreads();
    tile_row_idx = threadIdx.x / TileSizeX;
    tile_col_idx = threadIdx.x - tile_row_idx * TileSizeX;
    for (int32_t j = tile_row_idx; j < range_1; j += col_num_per_iter) {
      if (tile_col_idx < range_0) {
        out_tile_gmem[j * row + tile_col_idx] =
            tile_in_shmem[tile_col_idx][j ^ tile_col_idx];
      }
    }
    __syncthreads();
  }
}

template <typename T>
void batch_transpose(int32_t batch, int32_t row, int32_t col, const T *inp_ptr,
                     T *out_ptr, hipStream_t stream) {
  constexpr int32_t kTileSize = 32;

  const int32_t tile_num_in_dim0 = (row - 1) / kTileSize + 1;
  const int32_t tile_num_in_dim1 = (col - 1) / kTileSize + 1;
  const int32_t tile_per_sample = tile_num_in_dim0 * tile_num_in_dim1;
  const int32_t total_tile_num = batch * tile_per_sample;
  dim3 grid(total_tile_num >= kMaxGridDim ? kMaxGridDim : total_tile_num);
  if (row < 8 || col < 8) {
    constexpr int32_t kBlockSize = 64;
    dim3 block(kBlockSize);
    batch_transpose_kernel<T, kTileSize, kTileSize, kBlockSize>
        <<<grid, block, 0, stream>>>(
            total_tile_num, tile_num_in_dim0, tile_num_in_dim1, tile_per_sample,
            row, col, reinterpret_cast<void *>(const_cast<T *>(inp_ptr)),
            reinterpret_cast<void *>(out_ptr));
  } else {
    constexpr int32_t kBlockSize = 256;
    dim3 block(kBlockSize);
    batch_transpose_kernel<T, kTileSize, kTileSize, kBlockSize>
        <<<grid, block, 0, stream>>>(
            total_tile_num, tile_num_in_dim0, tile_num_in_dim1, tile_per_sample,
            row, col, reinterpret_cast<void *>(const_cast<T *>(inp_ptr)),
            reinterpret_cast<void *>(out_ptr));
  }
}

// instantiate
template void transpose_naive_2d<float>(const float *, float *, int, int, dim3,
                                        dim3, hipStream_t);
template void transpose_naive_2d<__half>(const __half *, __half *, int, int,
                                         dim3, dim3, hipStream_t);
template void batch_transpose<float>(int32_t, int32_t, int32_t, const float *,
                                     float *, hipStream_t);

template void batch_transpose<__half>(int32_t, int32_t, int32_t, const __half *,
                                      __half *, hipStream_t);
} // namespace kernel
} // namespace cuda
} // namespace brt
