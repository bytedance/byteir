#include "hip/hip_runtime.h"
#include "flash.h"
#include "flash_fwd_launch_template.h"
#include <iostream>

namespace brt {
namespace cuda {
namespace kernel {

// TODO: Switch back to handling bf16.
// void run_mha_fwd(Flash_fwd_params &params, hipStream_t stream) {
//   FWD_HEADDIM_SWITCH(params.d, [&] {
//     run_mha_fwd_<cutlass::half_t, kHeadDim>(params, stream);
//   });
// }

// void run_mha_fwd(Flash_fwd_params &params, hipStream_t stream) {
//     FP16_SWITCH(!params.is_bf16, [&] {
//         FWD_HEADDIM_SWITCH(params.d, [&] {
//             run_mha_fwd_<elem_type, kHeadDim>(params, stream);
//         });
//     });
// }

// for debug
void print_Qkv_params(Qkv_params &params) {
  std::cout << "q_batch_stride: " << params.q_batch_stride << std::endl;
  std::cout << "k_batch_stride: " << params.k_batch_stride << std::endl;
  std::cout << "v_batch_stride: " << params.v_batch_stride << std::endl;
  std::cout << "q_row_stride: " << params.q_row_stride << std::endl;
  std::cout << "k_row_stride: " << params.k_row_stride << std::endl;
  std::cout << "v_row_stride: " << params.v_row_stride << std::endl;
  std::cout << "q_head_stride: " << params.q_head_stride << std::endl;
  std::cout << "k_head_stride: " << params.k_head_stride << std::endl;
  std::cout << "v_head_stride: " << params.v_head_stride << std::endl;
  std::cout << "h: " << params.h << std::endl;
  std::cout << "h_k: " << params.h_k << std::endl;
  std::cout << "h_h_k_ratio: " << params.h_h_k_ratio << std::endl;
}

void print_Flash_fwd_params(Flash_fwd_params &params) {
  std::cout << "q_batch_stride: " << params.q_batch_stride << std::endl;
  std::cout << "k_batch_stride: " << params.k_batch_stride << std::endl;
  std::cout << "v_batch_stride: " << params.v_batch_stride << std::endl;
  std::cout << "q_row_stride: " << params.q_row_stride << std::endl;
  std::cout << "k_row_stride: " << params.k_row_stride << std::endl;
  std::cout << "v_row_stride: " << params.v_row_stride << std::endl;
  std::cout << "q_head_stride: " << params.q_head_stride << std::endl;
  std::cout << "k_head_stride: " << params.k_head_stride << std::endl;
  std::cout << "v_head_stride: " << params.v_head_stride << std::endl;
  std::cout << "h: " << params.h << std::endl;
  std::cout << "h_k: " << params.h_k << std::endl;
  std::cout << "h_h_k_ratio: " << params.h_h_k_ratio << std::endl;

  std::cout << "o_batch_stride: " << params.o_batch_stride << std::endl;
  std::cout << "o_row_stride: " << params.o_row_stride << std::endl;
  std::cout << "o_head_stride: " << params.o_head_stride << std::endl;
  std::cout << "b: " << params.b << std::endl;
  std::cout << "seqlen_q: " << params.seqlen_q << std::endl;
  std::cout << "seqlen_k: " << params.seqlen_k << std::endl;
  std::cout << "d: " << params.d << std::endl;
  std::cout << "seqlen_q_rounded: " << params.seqlen_q_rounded << std::endl;
  std::cout << "seqlen_k_rounded: " << params.seqlen_k_rounded << std::endl;
  std::cout << "d_rounded: " << params.d_rounded << std::endl;
  std::cout << "scale_softmax: " << params.scale_softmax << std::endl;
  std::cout << "scale_softmax_log2: " << params.scale_softmax_log2 << std::endl;
  std::cout << "p_dropout: " << params.p_dropout << std::endl;
  std::cout << "p_dropout_in_uint8_t: " << params.p_dropout_in_uint8_t
            << std::endl;
  std::cout << "rp_dropout: " << params.rp_dropout << std::endl;
  std::cout << "scale_softmax_rp_dropout: " << params.scale_softmax_rp_dropout
            << std::endl;
  std::cout << "is_bf16: " << params.is_bf16 << std::endl;
  std::cout << "is_causal: " << params.is_causal << std::endl;
}

void print_Flash_bwd_params(Flash_bwd_params &params) {
  std::cout << "q_batch_stride: " << params.q_batch_stride << std::endl;
  std::cout << "k_batch_stride: " << params.k_batch_stride << std::endl;
  std::cout << "v_batch_stride: " << params.v_batch_stride << std::endl;
  std::cout << "q_row_stride: " << params.q_row_stride << std::endl;
  std::cout << "k_row_stride: " << params.k_row_stride << std::endl;
  std::cout << "v_row_stride: " << params.v_row_stride << std::endl;
  std::cout << "q_head_stride: " << params.q_head_stride << std::endl;
  std::cout << "k_head_stride: " << params.k_head_stride << std::endl;
  std::cout << "v_head_stride: " << params.v_head_stride << std::endl;
  std::cout << "h: " << params.h << std::endl;
  std::cout << "h_k: " << params.h_k << std::endl;
  std::cout << "h_h_k_ratio: " << params.h_h_k_ratio << std::endl;

  std::cout << "o_batch_stride: " << params.o_batch_stride << std::endl;
  std::cout << "o_row_stride: " << params.o_row_stride << std::endl;
  std::cout << "o_head_stride: " << params.o_head_stride << std::endl;
  std::cout << "b: " << params.b << std::endl;
  std::cout << "seqlen_q: " << params.seqlen_q << std::endl;
  std::cout << "seqlen_k: " << params.seqlen_k << std::endl;
  std::cout << "d: " << params.d << std::endl;
  std::cout << "seqlen_q_rounded: " << params.seqlen_q_rounded << std::endl;
  std::cout << "seqlen_k_rounded: " << params.seqlen_k_rounded << std::endl;
  std::cout << "d_rounded: " << params.d_rounded << std::endl;
  std::cout << "scale_softmax: " << params.scale_softmax << std::endl;
  std::cout << "scale_softmax_log2: " << params.scale_softmax_log2 << std::endl;
  std::cout << "p_dropout: " << params.p_dropout << std::endl;
  std::cout << "p_dropout_in_uint8_t: " << params.p_dropout_in_uint8_t
            << std::endl;
  std::cout << "rp_dropout: " << params.rp_dropout << std::endl;
  std::cout << "scale_softmax_rp_dropout: " << params.scale_softmax_rp_dropout
            << std::endl;
  std::cout << "is_bf16: " << params.is_bf16 << std::endl;
  std::cout << "is_causal: " << params.is_causal << std::endl;

  std::cout << "do_batch_stride: " << params.do_batch_stride << std::endl;
  std::cout << "do_row_stride: " << params.do_row_stride << std::endl;
  std::cout << "do_head_stride: " << params.do_head_stride << std::endl;
  std::cout << "dq_batch_stride: " << params.dq_batch_stride << std::endl;
  std::cout << "dk_batch_stride: " << params.dk_batch_stride << std::endl;
  std::cout << "dv_batch_stride: " << params.dv_batch_stride << std::endl;
  std::cout << "dq_row_stride: " << params.dq_row_stride << std::endl;
  std::cout << "dk_row_stride: " << params.dk_row_stride << std::endl;
  std::cout << "dv_row_stride: " << params.dv_row_stride << std::endl;
  std::cout << "dq_head_stride: " << params.dq_head_stride << std::endl;
  std::cout << "dk_head_stride: " << params.dk_head_stride << std::endl;
  std::cout << "dv_head_stride: " << params.dv_head_stride << std::endl;
}

void run_mha(void *q_ptr, void *k_ptr, void *v_ptr, void *o_ptr,
             void *softmax_lse_ptr, void *softmax_ptr, void *rng_state_ptr,

             int32_t *cu_seqlens_q_ptr, int32_t *cu_seqlens_k_ptr,

             uint32_t q_batch_stride, uint32_t k_batch_stride,
             uint32_t v_batch_stride, uint32_t o_batch_stride,

             uint32_t q_row_stride, uint32_t k_row_stride,
             uint32_t v_row_stride, uint32_t o_row_stride,

             uint32_t q_head_stride, uint32_t k_head_stride,
             uint32_t v_head_stride, uint32_t o_head_stride,

             uint32_t b, uint32_t h, uint32_t h_k, uint32_t d,
             uint32_t d_rounded, float softmax_scale,

             uint32_t seqlen_q, uint32_t seqlen_k, uint32_t seqlen_q_rounded,
             uint32_t seqlen_k_rounded,

             float p_dropout, int is_causal, hipStream_t stream) {
  Flash_fwd_params params;
  // Reset the parameters
  memset(&params, 0, sizeof(params));

  // Set the pointers and strides.
  params.q_ptr = q_ptr;
  params.k_ptr = k_ptr;
  params.v_ptr = v_ptr;
  params.o_ptr = o_ptr;

  params.softmax_lse_ptr = softmax_lse_ptr;

  // All stride are in elements, not bytes.
  params.q_batch_stride = q_batch_stride;
  params.k_batch_stride = k_batch_stride;
  params.v_batch_stride = v_batch_stride;
  params.o_batch_stride = o_batch_stride;

  params.q_row_stride = q_row_stride;
  params.k_row_stride = k_row_stride;
  params.v_row_stride = v_row_stride;
  params.o_row_stride = o_row_stride;
  params.q_head_stride = q_head_stride;
  params.k_head_stride = k_head_stride;
  params.v_head_stride = v_head_stride;
  params.o_head_stride = o_head_stride;

  // Set the dimensions.
  params.b = b;
  params.h = h;
  params.h_k = h_k;
  params.h_h_k_ratio = h / h_k;
  params.seqlen_q = seqlen_q;
  params.seqlen_k = seqlen_k;
  params.seqlen_q_rounded = seqlen_q_rounded;
  params.seqlen_k_rounded = seqlen_k_rounded;
  params.d = d;
  params.d_rounded = d_rounded;
  params.is_causal = is_causal;

  // Set the different scale values.
  params.scale_softmax = softmax_scale;
  params.scale_softmax_log2 = softmax_scale * M_LOG2E;

  params.p_dropout = 1.f - p_dropout; // probability to keep
  params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
  params.rp_dropout = 1.f / params.p_dropout;
  params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
  params.is_bf16 = 0;
  params.cu_seqlens_q = cu_seqlens_q_ptr;
  params.cu_seqlens_k = cu_seqlens_k_ptr;
  params.p_ptr = softmax_ptr; // used for `return_softmax`.
  params.rng_state = static_cast<uint64_t *>(rng_state_ptr);

  // print_Flash_fwd_params(params);

  FP16_SWITCH(!params.is_bf16, [&] {
    FWD_HEADDIM_SWITCH(
        params.d, [&] { run_mha_fwd_<elem_type, kHeadDim>(params, stream); });
  });
}

void run_mha_bwd(void *q_ptr, void *k_ptr, void *v_ptr, void *o_ptr,
                 void *dout_ptr, void *dq_ptr, void *dk_ptr, void *dv_ptr,
                 int *cu_seqlens_q_ptr, int *cu_seqlens_k_ptr,
                 void *dq_accum_ptr, void *dk_accum_ptr, void *dv_accum_ptr,
                 void *softmax_lse_ptr, void *dsoftmax_sum_ptr,
                 void *rng_state_ptr,

                 uint32_t q_batch_stride, uint32_t k_batch_stride,
                 uint32_t v_batch_stride, uint32_t o_batch_stride,

                 uint32_t q_row_stride, uint32_t k_row_stride,
                 uint32_t v_row_stride, uint32_t o_row_stride,

                 uint32_t q_head_stride, uint32_t k_head_stride,
                 uint32_t v_head_stride, uint32_t o_head_stride,

                 uint32_t b, uint32_t h, uint32_t h_k, uint32_t d,
                 uint32_t d_rounded, float softmax_scale,

                 uint32_t seqlen_q, uint32_t seqlen_k,
                 uint32_t seqlen_q_rounded, uint32_t seqlen_k_rounded,

                 float p_dropout, int is_causal, hipStream_t stream) {
  Flash_bwd_params params;
  // Reset the parameters
  memset(&params, 0, sizeof(params));

  // Set the pointers and strides.
  params.q_ptr = q_ptr;
  params.k_ptr = k_ptr;
  params.v_ptr = v_ptr;
  params.o_ptr = o_ptr;

  params.dq_ptr = dq_ptr;
  params.dk_ptr = dk_ptr;
  params.dv_ptr = dv_ptr;
  params.do_ptr = dout_ptr;

  params.dq_accum_ptr = dq_accum_ptr;
  params.dk_accum_ptr = dk_accum_ptr;
  params.dv_accum_ptr = dv_accum_ptr;

  params.softmax_lse_ptr = softmax_lse_ptr;

  // All stride are in elements, not bytes.
  params.q_batch_stride = q_batch_stride;
  params.k_batch_stride = k_batch_stride;
  params.v_batch_stride = v_batch_stride;
  params.o_batch_stride = o_batch_stride;

  params.q_row_stride = q_row_stride;
  params.k_row_stride = k_row_stride;
  params.v_row_stride = v_row_stride;
  params.o_row_stride = o_row_stride;
  params.q_head_stride = q_head_stride;
  params.k_head_stride = k_head_stride;
  params.v_head_stride = v_head_stride;
  params.o_head_stride = o_head_stride;

  params.dq_batch_stride = q_batch_stride;
  params.dk_batch_stride = k_batch_stride;
  params.dv_batch_stride = v_batch_stride;
  params.do_batch_stride = o_batch_stride;

  params.dq_row_stride = q_row_stride;
  params.dk_row_stride = k_row_stride;
  params.dv_row_stride = v_row_stride;
  params.do_row_stride = o_row_stride;
  params.dq_head_stride = q_head_stride;
  params.dk_head_stride = k_head_stride;
  params.dv_head_stride = v_head_stride;
  params.do_head_stride = o_head_stride;

  // Set the dimensions.
  params.b = b;
  params.h = h;
  params.h_k = h_k;
  params.h_h_k_ratio = h / h_k;
  params.seqlen_q = seqlen_q;
  params.seqlen_k = seqlen_k;
  params.seqlen_q_rounded = seqlen_q_rounded;
  params.seqlen_k_rounded = seqlen_k_rounded;
  params.d = d;
  params.d_rounded = d_rounded;
  params.is_causal = is_causal;

  // Set the different scale values.
  params.scale_softmax = softmax_scale;
  params.scale_softmax_log2 = softmax_scale * M_LOG2E;

  params.p_dropout = 1.f - p_dropout; // probability to keep
  params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
  params.rp_dropout = 1.f / params.p_dropout;
  params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
  params.is_bf16 = 0;
  params.cu_seqlens_q = cu_seqlens_q_ptr;
  params.cu_seqlens_k = cu_seqlens_k_ptr;
  params.p_ptr = nullptr; // used for `return_softmax`, no use in bwd
  params.dsoftmax_sum = dsoftmax_sum_ptr;
  params.rng_state = static_cast<uint64_t *>(rng_state_ptr);

  // print_Flash_bwd_params(params);

  bool configure = false;
  FP16_SWITCH(!params.is_bf16, [&] {
    if (params.d <= 32) {
      run_mha_bwd_<elem_type, 32>(params, stream, configure);
    } else if (params.d <= 64) {
      run_mha_bwd_<elem_type, 64>(params, stream, configure);
    } else if (params.d <= 96) {
      run_mha_bwd_<elem_type, 96>(params, stream, configure);
    } else if (params.d <= 128) {
      run_mha_bwd_<elem_type, 128>(params, stream, configure);
    } else if (params.d <= 160) {
      run_mha_bwd_<elem_type, 160>(params, stream, configure);
    } else if (params.d <= 192) {
      run_mha_bwd_<elem_type, 192>(params, stream, configure);
    } else if (params.d <= 224) {
      run_mha_bwd_<elem_type, 224>(params, stream, configure);
    } else if (params.d <= 256) {
      run_mha_bwd_<elem_type, 256>(params, stream, configure);
    }
  });
}

} // namespace kernel
} // namespace cuda
} // namespace brt