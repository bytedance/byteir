#include "hip/hip_runtime.h"
//===- fill.cu ------------------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "./fill.h"

// TODO: move to common header
#define DIVUP(x, y) (((x) + (y)-1) / (y))

namespace brt {
namespace cuda {
namespace kernel {
template <typename T, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _Fill(T *output_data, T val, int32_t N) {
  int32_t id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output_data[id] = val;
      id += blockDim.x;
    }
  }
}

template <typename T>
void Fill(hipStream_t stream, T *output, T value, size_t count) {
  constexpr int maxThreadsPerBlock = 256;
  constexpr int maxElementsPerThread = 4;
  int blocksPerGrid =
      static_cast<int>(DIVUP(count, maxThreadsPerBlock * maxElementsPerThread));
  int32_t N = static_cast<int32_t>(count);
  _Fill<T, maxThreadsPerBlock, maxElementsPerThread>
      <<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(output, value, N);
}

#define INST(T) template void Fill<T>(hipStream_t, T *, T, size_t);

INST(float)
INST(int64_t)
INST(double)
INST(__half)
INST(int8_t)

#undef INST

} // namespace kernel
} // namespace cuda
} // namespace brt