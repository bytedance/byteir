#include "hip/hip_runtime.h"
//===- rng.cu -------------------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "./rng.h"
#include <atomic>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

// TODO: move to common header
#define DIVUP(x, y) (((x) + (y)-1) / (y))

namespace brt {
namespace cuda {
namespace kernel {
template <int NumElementsPerThread>
__global__ void _RngUniform(float *ptr, int32_t N, float base, float range,
                            size_t seed, size_t offset) {
  int32_t id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

  hiprandState_t state;

  // initialize local state with 2^67 * sequence + offset steps
  hiprand_init(seed,   /* seed */
              id,     /* sequence */
              offset, /* offset */
              &state);

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      float value = hiprand_uniform(&state);
      ptr[id] = base + value * range;
      id += blockDim.x;
    }
  }
}

namespace details {
struct GlobalRngState {
public:
  GlobalRngState() : seed_(0) { offset_.store(0); }

  size_t next(size_t n) { return offset_.fetch_add(n); }
  size_t seed() { return seed_; }

  static GlobalRngState *inst() {
    static GlobalRngState _;
    return &_;
  }

private:
  size_t seed_;
  std::atomic<size_t> offset_;
};
} // namespace details

void RngUniform(hipStream_t stream, float *ptr, size_t length, float low,
                float high) {
  constexpr int maxThreadsPerBlock = 256;
  constexpr int maxElementsPerThread = 4;
  int blocksPerGrid = static_cast<int>(
      DIVUP(length, maxThreadsPerBlock * maxElementsPerThread));
  int32_t N = static_cast<int32_t>(length);
  auto globalState = details::GlobalRngState::inst();
  size_t seed = globalState->seed();
  size_t offset = globalState->next(maxElementsPerThread);
  _RngUniform<maxElementsPerThread>
      <<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(
          ptr, N, low, high - low, seed, offset);
}

} // namespace kernel
} // namespace cuda
} // namespace brt