#include "hip/hip_runtime.h"
//===- rng.cu -------------------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "./rng.h"
#include <atomic>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

// TODO: move to common header
#define DIVUP(x, y) (((x) + (y)-1) / (y))

namespace brt {
namespace cuda {
namespace kernel {
template <int NumElementsPerThread>
__global__ void _RngUniformFloat(float *ptr, int32_t N, float base, float range,
                            size_t seed, size_t offset) {
  int32_t id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

  hiprandState_t state;

  // initialize local state with 2^67 * sequence + offset steps
  hiprand_init(seed,   /* seed */
              id,     /* sequence */
              offset, /* offset */
              &state);

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      float value = hiprand_uniform(&state);
      ptr[id] = base + value * range;
      id += blockDim.x;
    }
  }
}

template <int NumElementsPerThread>
__global__ void _RngUniformDouble(double *ptr, int32_t N, double base, double range,
                            size_t seed, size_t offset) {
  int32_t id = NumElementsPerThread * blockDim.x * blockIdx.x + threadIdx.x;

  hiprandState_t state;

  // initialize local state with 2^67 * sequence + offset steps
  hiprand_init(seed,   /* seed */
              id,     /* sequence */
              offset, /* offset */
              &state);

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      double value = hiprand_uniform_double(&state);
      ptr[id] = base + value * range;
      id += blockDim.x;
    }
  }
}

namespace details {
struct GlobalRngState {
public:
  GlobalRngState() : seed_(0) { offset_.store(0); }

  size_t next(size_t n) { return offset_.fetch_add(n); }
  size_t seed() { return seed_; }

  static GlobalRngState *inst() {
    static GlobalRngState _;
    return &_;
  }

private:
  size_t seed_;
  std::atomic<size_t> offset_;
};
} // namespace details

template <typename InputTy>
void RngUniform(hipStream_t stream, InputTy *ptr, size_t length, InputTy low,
                InputTy high);

template <>
void RngUniform<float>(hipStream_t stream, float *ptr, size_t length, float low,
                float high) {
  constexpr int maxThreadsPerBlock = 256;
  constexpr int maxElementsPerThread = 4;
  int blocksPerGrid = static_cast<int>(
      DIVUP(length, maxThreadsPerBlock * maxElementsPerThread));
  int32_t N = static_cast<int32_t>(length);
  auto globalState = details::GlobalRngState::inst();
  size_t seed = globalState->seed();
  size_t offset = globalState->next(maxElementsPerThread);
  _RngUniformFloat<maxElementsPerThread>
      <<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(
          ptr, N, low, high - low, seed, offset);
}

template <>
void RngUniform<double>(hipStream_t stream, double *ptr, size_t length, double low,
                double high) {
  constexpr int maxThreadsPerBlock = 256;
  constexpr int maxElementsPerThread = 4;
  int blocksPerGrid = static_cast<int>(
      DIVUP(length, maxThreadsPerBlock * maxElementsPerThread));
  int32_t N = static_cast<int32_t>(length);
  auto globalState = details::GlobalRngState::inst();
  size_t seed = globalState->seed();
  size_t offset = globalState->next(maxElementsPerThread);
  _RngUniformDouble<maxElementsPerThread>
      <<<blocksPerGrid, maxThreadsPerBlock, 0, stream>>>(
          ptr, N, low, high - low, seed, offset);
}

} // namespace kernel
} // namespace cuda
} // namespace brt