#include "hip/hip_runtime.h"
//===- index_put.cu -------------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

#include "./index_put.h"
#include "brt/backends/cuda/device/common/cuda_call.h"
#include <algorithm>

namespace brt {
namespace cuda {
namespace kernel {

// a native_input_put putting entire inner_loop (feature dim) based outer_loop
// (embedding dim)
template <typename T, bool Accum>
__global__ void naive_index_put_kernel(T *inout, const int64_t *indices,
                                       const T *update,
                                       const int feature_bound) {
  int out_offset = indices[blockIdx.x];
  for (int idx = threadIdx.x; idx < feature_bound; idx += blockDim.x) {
    int in_idx = blockIdx.x * feature_bound + idx;
    int out_idx = out_offset * feature_bound + idx;
    T value = update[in_idx];
    if (Accum) {
      atomicAdd((T *)(inout + out_idx), value);
    } else {
      inout[out_idx] = value;
    }
  }
}

template <typename T, bool Accum>
void index_put(const T *input, const int64_t *indices, const T *update,
               T *output, const int index_count, const int feature_bound,
               const int size, hipStream_t stream) {
  BRT_CUDA_CHECK(hipMemcpyAsync(output, input, size * sizeof(T),
                                 hipMemcpyDeviceToDevice, stream));
  dim3 grid = index_count;
  dim3 block = std::min(256, feature_bound);
  naive_index_put_kernel<T, Accum>
      <<<grid, block, 0, stream>>>(output, indices, update, feature_bound);
}

template void index_put<float, true>(const float *, const int64_t *,
                                     const float *, float *, const int,
                                     const int, const int, hipStream_t);

} // namespace kernel
} // namespace cuda
} // namespace brt
