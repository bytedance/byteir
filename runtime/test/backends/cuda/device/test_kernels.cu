
#include <hip/hip_runtime.h>
//===- test_kernels.cu ----------------------------------------*--- C++ -*-===//
//
// Copyright 2022 ByteDance Ltd. and/or its affiliates. All rights reserved.
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//    http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
//
//===----------------------------------------------------------------------===//

namespace brt {
namespace test {
// TODO move this kernel to another separate file
__global__ void test_kernel(const float *input, float *output, int n,
                            float val) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    output[i] = input[i] + val;
  }
}
} // namespace test
} // namespace brt
