
#include <hip/hip_runtime.h>
extern "C" __global__ void nvrtc_add_kernel(const float* input, float* output, int n, float val) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    output[i] = input[i]+ val;
  }
}